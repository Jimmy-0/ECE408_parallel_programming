#include "hip/hip_runtime.h"
#define M_TILE_H 16 // Height of input M tiles (Y dimension)
#define M_TILE_W 64 // Width of input M tiles (X dimension) 
#define N_TILE_H 64 // Height of input N tiles (Y dimension)
#define N_TILE_W 16 // Weight of input N tiles (X dimension)

__global__ void sgemm(float* M, float* N, float* P, int HeiM, int WidM, int WidN) { 
    
    __shared__ float Mds[M_TILE_H][M_TILE_W]; // constant row, loop varying column
    __shared__ float Nds[N_TILE_H][N_TILE_W]; // constant column, loop varying row

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

 // Identify the row and column of the P element to work on
    int Row = by * M_TILE_H + ty;
    int Col = bx * N_TILE_W + tx;

 float Pvalue = 0;
 // Loop over the M and N tiles required to compute P element
    for (int m = 0; m < (WidM - 1)/ M_TILE_W + 1; ++m) {

 // Collaborative load of M and N tiles into shared memory
        if(Row < HeiM) { Mds[ty][tx] = M[Row * WidM + (m*M_TILE_W)+tx]} 
            else {Mds[ty][tx] = 0.0;
            }
        if(Col < WidN) {Nds[ty][tx] = N[(m * N_TILE_H + ty) * WidN+Col]} 
            else {Nds[ty][tx] = 0.0;
            }
        __syncthreads();

        if (Row < HeiM && Col < WidN) {
            for (int k = 0; k < M_TILE_W; ++k) {
                Pvalue += Mdsp[ty][k] * Nds [k][tx]
                }
        }
 __syncthreads();
    }
    if (Row < HeiM && Col < WidN) P[Row*Width + Col] = Pvalue;
}
